#include "hip/hip_runtime.h"
#include "newtonFractal.cuh"
#include "structs.h"
#include "hip/hip_runtime.h"
#include "cuda_error_helper.hpp"


__global__ void initialiseFramebuffer(unsigned char* frameBuffer, int width, int height) {
	// Initializes framebuffer to black
	unsigned int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

	if (threadIndex >= 4 * width * height) {
		return;
	}

	if (threadIndex % 4 == 3) {
		frameBuffer[threadIndex] = 255;
	}
	else {
		frameBuffer[threadIndex] = 0;
	}
}

__global__ void dummyKernel(unsigned char* frameBuffer, int width, int heigth) {
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadIdx.x < 128) {
		frameBuffer[4 * index] = 0;
		frameBuffer[4 * index + 1] = 0;
		frameBuffer[4 * index + 2] = 0;
	}
}

__global__ void newtonFractal1(unsigned const int width, unsigned const int height, 
	const float scale,
	unsigned char* frameBuffer) {
	unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
	// Initial guess
	hipFloatComplex z = make_hipComplex(scale * (-1.0 + 2.0 * ((float)(index%width) / (float)width)), scale * (-1.0 + 2.0 * floorf(index / width) / (float)height));
	// Do 10 steps of newton
	for (unsigned int i = 0; i < 40; i++) {
		z = hipCsubf(z, hipCdivf(hipCsubf(1000000000*hipCmulf(z,hipCmulf(z,z)), make_hipFloatComplex(1,0)), 3000000000 * hipCmulf(z,z)));
	}
	// Find closest result
	int result = -1;
	hipFloatComplex zeros[3] = {
		0.001 * make_hipFloatComplex(1.0f,0.0f),
		0.001 * make_hipFloatComplex(-0.5f, 0.8660254037844f),
		0.001 * make_hipFloatComplex(-0.5f, -0.8660254037844f) };
	for (unsigned int i = 0; i < 3; i++) {
		if (cuCnormf(hipCsubf(z, zeros[i])) < 1e-8) {
			result = i;
		}
	}
	if (result != -1) {
		frameBuffer[4 * index + 0] = 0;
		frameBuffer[4 * index + 1] = 0;
		frameBuffer[4 * index + 2] = 0;
		frameBuffer[4 * index + result] = 255;
	}
}

std::vector<unsigned char> makefb(unsigned int width, unsigned int height) {
	unsigned char* frameBuffer = new unsigned char[width * height * 4];
	unsigned char* device_frameBuffer;
	checkCudaErrors(hipMalloc(&device_frameBuffer, width * height * 4 * sizeof(unsigned char)));
	const unsigned int initialisationBlockSize = 256;
	 
	unsigned int blockCountFrameBuffer = ((width * height * 4) / initialisationBlockSize);
	initialiseFramebuffer<<<blockCountFrameBuffer, initialisationBlockSize>>> (device_frameBuffer, width, height);
	checkCudaErrors(hipDeviceSynchronize());
	
	unsigned int blockSize = 256;
	unsigned int gridSize = width * height / blockSize;
	//dummyKernel<<<gridSize, blockSize>>> (device_frameBuffer, width, height);
	//checkCudaErrors(hipDeviceSynchronize())
	newtonFractal1<<<gridSize, blockSize>>>(width, height,
		0.00001f,
		device_frameBuffer
		);
	checkCudaErrors(hipDeviceSynchronize());
	std::vector<unsigned char> outputFramebuffer(frameBuffer, frameBuffer + (width * height * 4));
	checkCudaErrors(hipMemcpy(outputFramebuffer.data(), device_frameBuffer, width * height * 4 * sizeof(unsigned char), hipMemcpyDeviceToHost));
	checkCudaErrors(hipDeviceSynchronize());
	hipDeviceReset();
	return outputFramebuffer;
}
